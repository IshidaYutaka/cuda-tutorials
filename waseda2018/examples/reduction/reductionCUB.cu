#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "cub/hipcub/hipcub.hpp"

long long getCurrentTime()
{
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long  microseconds = te.tv_sec*1000000LL + te.tv_usec; 
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

__global__ void reduce(int *A, int *sum, int N)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
	 i < N;
	 i += blockDim.x * gridDim.x) {
	atomicAdd(sum, A[i]);
    }
}

int ReduceCPU(int *A, int N, double *cpuTime)
{
    long long startTime = getCurrentTime();
    int sum = 0;
    for (int i = 0; i < N; i++) {
	sum += A[i];
    }
    *cpuTime = (double)(getCurrentTime() - startTime) / 1000000;
    return sum;
}

int ReduceGPU(int *A, int N, double *gpuOverallTime, double *gpuKernelTime) {
    long long startTime = getCurrentTime();
    
    int threads = 512;
    int blocks = min((N + threads - 1) / threads, 1024);

    int *S = (int*)malloc(sizeof(int) * 1);
    int *dA;
    int *dSum;

    CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));
    CudaSafeCall(hipMalloc(&dSum, sizeof(int) * 1));
    
    CudaSafeCall(hipMemcpy(dA, A, N * sizeof (int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemset(dSum, 0, sizeof (int)));

    // Determine temporary device storage requirements
    size_t temp_storage_bytes = 0;
    int* temp_storage=NULL;
    int init = 0;
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, dA, dSum, N, hipcub::Sum(), init);
    // Allocate temporary storage
    hipMalloc(&temp_storage, temp_storage_bytes);
    hipDeviceSynchronize();
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    hipEventRecord(start);
    // Run reduction
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, dA, dSum, N, hipcub::Sum(), init);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    CudaSafeCall(hipGetLastError());

    CudaSafeCall(hipDeviceSynchronize());
    CudaSafeCall(hipMemcpy(S, dSum, 1 * sizeof (int), hipMemcpyDeviceToHost));

    *gpuOverallTime = (double)(getCurrentTime() - startTime) / 1000000;
    
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    *gpuKernelTime = msec / 1000;

    hipFree(dA);
    hipFree(dSum);

    return *S;
}

int
main(int argc, char **argv)
{

    if (argc != 2) {
	printf("Usage: ./reduce repeat\n");
	exit(0);
    }
    int REPEATS = atoi(argv[1]);
    
    for (int repeat = 0; repeat < REPEATS; repeat++) {
	printf("[Iteration %d]\n", repeat);
	for (int N = 1024; N < 256 * 1024 * 1024; N = N * 2) {
	    int* A = NULL;
	    double cpuTime = 0.0;
	    double gpuOverallTime = 0.0;
	    double gpuKernelTime = 0.0;
	
	    A = (int*)malloc(sizeof(int) * N);
	    
	    for (int i = 0; i < N; i++) {
		A[i] = i;
	    }

	    // CPU version	    
	    int expected = ReduceCPU(A, N, &cpuTime);

	    // GPU version
	    int computed = ReduceGPU(A, N, &gpuOverallTime, &gpuKernelTime);
	    	    
	    if (computed == expected) {
		float GB = (float)(N * 4) / (1024 * 1024 * 1024);
		printf ("\tVERIFIED, %d, CPU (%lf sec) %lf GB/s, GPU (Overall: %lf sec) %lf GB/s, GPU (Kernel: %lf sec) %lf GB/s\n", 4*N, cpuTime, GB / cpuTime, gpuOverallTime, GB / gpuOverallTime, gpuKernelTime, GB / gpuKernelTime);
	    } else {
		printf ("\tFAILED, %d, computed: %d, excepted %u\n", 4*N, computed, expected);
	    }
	    
	    free(A);

	}
    }
}    
