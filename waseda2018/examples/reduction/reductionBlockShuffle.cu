#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long long getCurrentTime() {
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec; 
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

__inline__ __device__ int warpReduceSum(int val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
#if CUDA_VERSION >= 9000
      val += __shfl_down_sync(0xffffffff, val, offset);
#else
      val += __shfl_down(val, offset);
#endif	
    }
    return val;
}

__inline__ __device__ int blockReduceSum(int val) {
    static __shared__ int shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = warpReduceSum(val);
    if (lane == 0) shared[wid] = val;
    __syncthreads();
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if (wid==0) val = warpReduceSum(val);
    return val;
}

__global__ void reduce(int *A, int *sum, int N)
{
    int val = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
	 i < N;
	 i+= blockDim.x * gridDim.x) {
	val += A[i];
    }
    int valPerBlock = blockReduceSum(val);
    if (threadIdx.x == 0) {
	sum[blockIdx.x] = valPerBlock;
    }
}

int ReduceCPU(int *A, int N, double *cpuTime)
{
    long long startTime = getCurrentTime();
    int sum = 0;
    for (int i = 0; i < N; i++) {
	sum += A[i];
    }
    *cpuTime = (double)(getCurrentTime() - startTime) / 1000000;
    return sum;
}

int ReduceGPU(int *A, int N, double *gpuOverallTime, double *gpuKernelTime)
{
    long long startTime = getCurrentTime();
    
    int threads = 512;
    int blocks = min((N + threads - 1) / threads, 1024);

    int *S = (int*)malloc(sizeof(int) * 1);
    int *dA;
    int *dSum;

    // Allocate memory on the device
    CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));
    CudaSafeCall(hipMalloc(&dSum, sizeof(int) * 1024));

    // Copy the data from the host to the device
    CudaSafeCall(hipMemcpy(dA, A, N * sizeof (int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemset(dSum, 0, sizeof (int) * 1024));
    
    hipEvent_t start, stop;
    CudaSafeCall(hipEventCreate(&start));
    CudaSafeCall(hipEventCreate(&stop));

    // Launch the kernel
    CudaSafeCall(hipEventRecord(start));
    reduce<<<blocks, threads>>>(dA, dSum, N);
    reduce<<<1, 1024>>>(dSum, dSum, 1024);
    CudaSafeCall(hipEventRecord(stop));
    CudaSafeCall(hipEventSynchronize(stop));
    CudaSafeCall(hipDeviceSynchronize());

    // Copy back the data from the host
    CudaSafeCall(hipMemcpy(S, dSum, 1 * sizeof (int), hipMemcpyDeviceToHost));

    *gpuOverallTime = (double)(getCurrentTime() - startTime) / 1000000;
    
    float msec = 0;
    CudaSafeCall(hipEventElapsedTime(&msec, start, stop));
    *gpuKernelTime = msec / 1000;

    CudaSafeCall(hipFree(dA));
    CudaSafeCall(hipFree(dSum));

    return *S;
}

int
main(int argc, char **argv)
{

    if (argc != 2) {
	printf("Usage: ./reduce repeat\n");
	exit(0);
    }
    int REPEATS = atoi(argv[1]);
    
    for (int repeat = 0; repeat < REPEATS; repeat++) {
	printf("[Iteration %d]\n", repeat);
	for (int N = 1024; N < 256 * 1024 * 1024; N = N * 2) {
	    int* A = NULL;
	    double cpuTime = 0.0;
	    double gpuOverallTime = 0.0;
	    double gpuKernelTime = 0.0;
	
	    A = (int*)malloc(sizeof(int) * N);
	    
	    for (int i = 0; i < N; i++) {
		A[i] = i;
	    }

	    // CPU version	    
	    int expected = ReduceCPU(A, N, &cpuTime);

	    // GPU version
	    int computed = ReduceGPU(A, N, &gpuOverallTime, &gpuKernelTime);
	    	    
	    if (computed == expected) {
		float GB = (float)(N * 4) / (1024 * 1024 * 1024);
		printf ("\tVERIFIED, %d, CPU (%lf sec) %lf GB/s, GPU (Overall: %lf sec) %lf GB/s, GPU (Kernel: %lf sec) %lf GB/s\n", 4*N, cpuTime, GB / cpuTime, gpuOverallTime, GB / gpuOverallTime, gpuKernelTime, GB / gpuKernelTime);
	    } else {
		printf ("\tFAILED, %d, computed: %d, excepted %u\n", 4*N, computed, expected);
	    }
	    
	    free(A);

	}
    }
}    
