#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

long long getCurrentTime()
{
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec;
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

__device__ int tile_reduce(cg::thread_block_tile<32> g, int val)
{
    for (int i = g.size() / 2; i > 0; i /= 2) {
	val += g.shfl_down(val, i);
    }
    return val;
}

__global__ void reduce(int *A, int *sum, int N)
{
    // Handle to thread block group
    cg::thread_block block = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int val = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
	 i < N;
	 i += blockDim.x * gridDim.x) {
	val += A[i];
    }

    cg::thread_block_tile<32> g = cg::tiled_partition<32>(cg::this_thread_block());
    int valPerWarp = tile_reduce(g, val);

    if (g.thread_rank() == 0) {
	atomicAdd(sum, valPerWarp);
    }
}

int ReduceCPU(int *A, int N, double *cpuTime)
{
    long long startTime = getCurrentTime();
    int sum = 0;
    for (int i = 0; i < N; i++) {
	sum += A[i];
    }
    *cpuTime = (double)(getCurrentTime() - startTime) / 1000000;
    return sum;
}

int ReduceGPU(int *A, int N, double *gpuOverallTime, double *gpuKernelTime)
{
    long long startTime = getCurrentTime();

    int threads = 512;
    int numBlocksPerSM;
    int numSms = 80; // Volta
    CudaSafeCall(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, reduce, threads, threads*sizeof(int)));
    
    int blocks = min((N + threads - 1) / threads, numBlocksPerSM * numSms);

    int *S = (int*)malloc(sizeof(int) * 1);
    int *dA;
    int *dSum;

    // Allocate memory on the device
    CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));
    CudaSafeCall(hipMalloc(&dSum, sizeof(int) * 1));

    // Copy the data from the host to the device
    CudaSafeCall(hipMemcpy(dA, A, N * sizeof (int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemset(dSum, 0, sizeof (int)));
    
    hipEvent_t start, stop;
    CudaSafeCall(hipEventCreate(&start));
    CudaSafeCall(hipEventCreate(&stop));

    int size = N;
    void *kernelArgs[] = {(void*)&dA, (void*)&dSum, (void*)&size};
    // Launch the kernel
    CudaSafeCall(hipEventRecord(start));
    CudaSafeCall(hipLaunchCooperativeKernel((void*)reduce, blocks, threads, kernelArgs));
    CudaSafeCall(hipEventRecord(stop));
    CudaSafeCall(hipEventSynchronize(stop));
    CudaSafeCall(hipDeviceSynchronize());

    // Copy back the data from the host
    CudaSafeCall(hipMemcpy(S, dSum, 1 * sizeof (int), hipMemcpyDeviceToHost));

    // Compute the performance numbers
    *gpuOverallTime = (double)(getCurrentTime() - startTime) / 1000000;
    float msec = 0;
    CudaSafeCall(hipEventElapsedTime(&msec, start, stop));
    *gpuKernelTime = msec / 1000;

    // Cleanup
    CudaSafeCall(hipFree(dA));
    CudaSafeCall(hipFree(dSum));

    return *S;
}

int main(int argc, char **argv)
{

    if (argc != 2) {
	printf("Usage: ./reduce repeat\n");
	exit(0);
    }
    int REPEATS = atoi(argv[1]);
    
    for (int repeat = 0; repeat < REPEATS; repeat++) {
	printf("[Iteration %d]\n", repeat);
	for (int N = 1024; N < 256 * 1024 * 1024; N = N * 2) {
	    int* A = NULL;
	    double cpuTime = 0.0;
	    double gpuOverallTime = 0.0;
	    double gpuKernelTime = 0.0;
	
	    A = (int*)malloc(sizeof(int) * N);
	    
	    for (int i = 0; i < N; i++) {
		A[i] = i;
	    }

	    // CPU version	    
	    int expected = ReduceCPU(A, N, &cpuTime);

	    // GPU version
	    int computed = ReduceGPU(A, N, &gpuOverallTime, &gpuKernelTime);
	    	    
	    if (computed == expected) {
		float GB = (float)(N * 4) / (1024 * 1024 * 1024);
		printf ("\tVERIFIED, %d, CPU (%lf sec) %lf GB/s, GPU (Overall: %lf sec) %lf GB/s, GPU (Kernel: %lf sec) %lf GB/s\n", 4*N, cpuTime, GB / cpuTime, gpuOverallTime, GB / gpuOverallTime, gpuKernelTime, GB / gpuKernelTime);
	    } else {
		printf ("\tFAILED, %d, computed: %d, excepted %u\n", 4*N, computed, expected);
	    }
	    
	    free(A);

	}
    }
}    
