#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long long getCurrentTime()
{
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec;
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err ) {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

int main()
{
    for (int N = 1024; N < 256 * 1024* 1024; N = N * 2) {
        int *pageA, *pageB, *pinnedA, *pinnedB;
        int *dA;

        // Allocate pageable memory on the host
        pageA = (int*)malloc(sizeof(int) * N);
        pageB = (int*)malloc(sizeof(int) * N);

        // Allocate pinned memory on the
        CudaSafeCall(hipHostMalloc((void**)&pinnedA, sizeof(int) * N, hipHostMallocDefault));
        CudaSafeCall(hipHostMalloc((void**)&pinnedB, sizeof(int) * N, hipHostMallocDefault));

        // Allocate memory on the device
        CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));

        // Initilize data
        for (int i = 0; i < N; i++) {
            pageA[i] = i;
            pinnedA[i] = i;
        }

        int error = 0;

        hipEvent_t startH2DPage, endH2DPage;
        hipEvent_t startD2HPage, endD2HPage;
        hipEvent_t startH2DPinned, endH2DPinned;
        hipEvent_t startD2HPinned, endD2HPinned;

        // Measuring H2D/D2H performance (Pageable Memory)
        {
            CudaSafeCall(hipEventCreate(&startH2DPage));
            CudaSafeCall(hipEventCreate(&endH2DPage));
            CudaSafeCall(hipEventCreate(&startD2HPage));
            CudaSafeCall(hipEventCreate(&endD2HPage));

            // H2D
            CudaSafeCall(hipEventRecord(startH2DPage));
            CudaSafeCall(hipMemcpy(dA, pageA, sizeof(int) * N, hipMemcpyHostToDevice));
            CudaSafeCall(hipEventRecord(endH2DPage));
            CudaSafeCall(hipEventSynchronize(endH2DPage));

            // D2H
            CudaSafeCall(hipEventRecord(startD2HPage));
            CudaSafeCall(hipMemcpy(pageB, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
            CudaSafeCall(hipEventRecord(endD2HPage));
            CudaSafeCall(hipEventSynchronize(endD2HPage));

            error = 0;
            for (int i = 0; i < N; i++) {
                if (pageA[i] != pageB[i]) {
                    error++;
                }
            }
        }

        // Measuring H2D/D2H performance (Pinned Memory)
        {
            CudaSafeCall(hipEventCreate(&startH2DPinned));
            CudaSafeCall(hipEventCreate(&endH2DPinned));
            CudaSafeCall(hipEventCreate(&startD2HPinned));
            CudaSafeCall(hipEventCreate(&endD2HPinned));

            // H2D
            CudaSafeCall(hipEventRecord(startH2DPinned));
            CudaSafeCall(hipMemcpy(dA, pinnedA, sizeof(int) * N, hipMemcpyHostToDevice));
            CudaSafeCall(hipEventRecord(endH2DPinned));
            CudaSafeCall(hipEventSynchronize(endH2DPinned));

            // D2H
            CudaSafeCall(hipEventRecord(startD2HPinned));
            CudaSafeCall(hipMemcpy(pinnedB, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
            CudaSafeCall(hipEventRecord(endD2HPinned));
            CudaSafeCall(hipEventSynchronize(endD2HPinned));

            for (int i = 0; i < N; i++) {
                if (pinnedA[i] != pinnedB[i]) {
                    error++;
                }
            }
        }

        if (!error) {
            float h2dpage, d2hpage, h2dpinned, d2hpinned;
            CudaSafeCall(hipEventElapsedTime(&h2dpage, startH2DPage, endH2DPage));
            CudaSafeCall(hipEventElapsedTime(&d2hpage, startD2HPage, endD2HPage));
            CudaSafeCall(hipEventElapsedTime(&h2dpinned, startH2DPinned, endH2DPinned));
            CudaSafeCall(hipEventElapsedTime(&d2hpinned, startD2HPinned, endD2HPinned));
            printf("Size: %lu bytes, H2D Page: %lf msec, D2H Page: %lf msec, H2D Pinned: %lf msec, D2H Pinned: %lf msec\n", N * sizeof(int), h2dpage, d2hpage, h2dpinned, d2hpinned);
        }
    }

    return 0;
}
