#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long long getCurrentTime() {
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec; 
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

int
main()
{


    for (int N = 1024; N < 256 * 1024* 1024; N = N * 2) {
	int *pageA, *pageB, *pinnedA, *pinnedB;
	int *dA;

	//
	pageA = (int*)malloc(sizeof(int) * N);
	pageB = (int*)malloc(sizeof(int) * N);
	CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));

	//
	CudaSafeCall(hipHostMalloc((void**)&pinnedA, sizeof(int) * N, hipHostMallocDefault));
	CudaSafeCall(hipHostMalloc((void**)&pinnedB, sizeof(int) * N, hipHostMallocDefault));
	
	//
	for (int i = 0; i < N; i++) {
	    pageA[i] = i;
	    pinnedA[i] = i;
	}

	//
	hipEvent_t startH2DPage, endH2DPage;
	hipEvent_t startD2HPage, endD2HPage;
	CudaSafeCall(hipEventCreate(&startH2DPage));
	CudaSafeCall(hipEventCreate(&endH2DPage));
	CudaSafeCall(hipEventCreate(&startD2HPage));
	CudaSafeCall(hipEventCreate(&endD2HPage));

	CudaSafeCall(hipEventRecord(startH2DPage));	
	CudaSafeCall(hipMemcpy(dA, pageA, sizeof(int) * N, hipMemcpyHostToDevice));
	CudaSafeCall(hipEventRecord(endH2DPage));
	CudaSafeCall(hipEventSynchronize(endH2DPage));

	CudaSafeCall(hipEventRecord(startD2HPage));	
	CudaSafeCall(hipMemcpy(pageB, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
	CudaSafeCall(hipEventRecord(endD2HPage));
	CudaSafeCall(hipEventSynchronize(endD2HPage));
       
	int error = 0;
	for (int i = 0; i < N; i++) {
	    if (pageA[i] != pageB[i]) {
		error++;
	    }
	}

	//
	hipEvent_t startH2DPinned, endH2DPinned;
	hipEvent_t startD2HPinned, endD2HPinned;
	CudaSafeCall(hipEventCreate(&startH2DPinned));
	CudaSafeCall(hipEventCreate(&endH2DPinned));
	CudaSafeCall(hipEventCreate(&startD2HPinned));
	CudaSafeCall(hipEventCreate(&endD2HPinned));

	CudaSafeCall(hipEventRecord(startH2DPinned));		
	CudaSafeCall(hipMemcpy(dA, pinnedA, sizeof(int) * N, hipMemcpyHostToDevice));
	CudaSafeCall(hipEventRecord(endH2DPinned));
	CudaSafeCall(hipEventSynchronize(endH2DPinned));

	CudaSafeCall(hipEventRecord(startD2HPinned));		
	CudaSafeCall(hipMemcpy(pinnedB, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
	CudaSafeCall(hipEventRecord(endD2HPinned));
	CudaSafeCall(hipEventSynchronize(endD2HPinned));

	for (int i = 0; i < N; i++) {
	    if (pinnedA[i] != pinnedB[i]) {
		error++;
	    }
	}
	
	if (!error) {
	    float h2dpage, d2hpage, h2dpinned, d2hpinned;
	    CudaSafeCall(hipEventElapsedTime(&h2dpage, startH2DPage, endH2DPage));
	    CudaSafeCall(hipEventElapsedTime(&d2hpage, startD2HPage, endD2HPage));
	    CudaSafeCall(hipEventElapsedTime(&h2dpinned, startH2DPinned, endH2DPinned));
	    CudaSafeCall(hipEventElapsedTime(&d2hpinned, startD2HPinned, endD2HPinned));
	    printf("Size: %lu bytes, H2D Page: %lf msec, D2H Page: %lf msec, H2D Pinned: %lf msec, D2H Pinned: %lf msec\n", N * sizeof(int), h2dpage, d2hpage, h2dpinned, d2hpinned);
	}
    }
    
    return 0;
}    
