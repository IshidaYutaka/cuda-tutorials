#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long long getCurrentTime() {
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec; 
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

__global__ void assign(int *A, int *B)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    A[i] = B[i];
}

int
main()
{
    int *A, *B;
    int *dA, *dB;
    int N = 256 * 1024 * 1024;
    long long startCudaMalloc, endCudaMalloc;
    long long startCudaMemcpyH2D, endCudaMemcpyH2D;
    long long startCudaKernel, endCudaKernel;
    long long startCudaMemcpyD2H, endCudaMemcpyD2H;
    hipEvent_t startCudaMallocEvent, endCudaMallocEvent;
    hipEvent_t startCudaMemcpyH2DEvent, endCudaMemcpyH2DEvent;
    hipEvent_t startCudaKernelEvent, endCudaKernelEvent;
    hipEvent_t startCudaMemcpyD2HEvent, endCudaMemcpyD2HEvent;
    float msecTmp;
    
    CudaSafeCall(hipEventCreate(&startCudaMallocEvent));
    CudaSafeCall(hipEventCreate(&endCudaMallocEvent));
    CudaSafeCall(hipEventCreate(&startCudaMemcpyH2DEvent));
    CudaSafeCall(hipEventCreate(&endCudaMemcpyH2DEvent));
    CudaSafeCall(hipEventCreate(&startCudaKernelEvent));
    CudaSafeCall(hipEventCreate(&endCudaKernelEvent));
    CudaSafeCall(hipEventCreate(&startCudaMemcpyD2HEvent));
    CudaSafeCall(hipEventCreate(&endCudaMemcpyD2HEvent));
            
    // Step 1: Allocate memory on the host (use malloc)
    A = (int*)malloc(sizeof(int) * N);
    B = (int*)malloc(sizeof(int) * N);

    for (int i = 0; i < N; i++) {
	B[i] = i;
    }

    // Step 2: Allocate memory on the device (use cudaMalloc)
    startCudaMalloc = getCurrentTime();
    {
	CudaSafeCall(hipEventRecord(startCudaMallocEvent));
	CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));
	CudaSafeCall(hipMalloc(&dB, sizeof(int) * N));
	CudaSafeCall(hipEventRecord(endCudaMallocEvent));
	CudaSafeCall(hipEventSynchronize(endCudaMallocEvent));
    }
    endCudaMalloc = getCurrentTime();
    CudaSafeCall(hipEventElapsedTime(&msecTmp, startCudaMallocEvent, endCudaMallocEvent));
    printf("cudaMalloc, getCurrentTime = %lf msec, hipEventElapsedTime = %lf msec\n", (float)(endCudaMalloc-startCudaMalloc)/1000, msecTmp);
    
    // Step 3: Copy the host data to the device (use cudaMemcpy) 
    startCudaMemcpyH2D = getCurrentTime();
    {
	CudaSafeCall(hipEventRecord(startCudaMemcpyH2DEvent));
	CudaSafeCall(hipMemcpy(dA, A, sizeof(int) * N, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dB, B, sizeof(int) * N, hipMemcpyHostToDevice));
	CudaSafeCall(hipEventRecord(endCudaMemcpyH2DEvent));
	CudaSafeCall(hipEventSynchronize(endCudaMemcpyH2DEvent));
    }
    endCudaMemcpyH2D = getCurrentTime();
    CudaSafeCall(hipEventElapsedTime(&msecTmp, startCudaMemcpyH2DEvent, endCudaMemcpyH2DEvent));
    printf("cudaMemcpy, getCurrentTime = %lf msec, hipEventElapsedTime = %lf msec\n", (float)(endCudaMemcpyH2D-startCudaMemcpyH2D)/1000, msecTmp);
    
    // Step 4: Launch the kernel
    startCudaKernel = getCurrentTime();
    {
	CudaSafeCall(hipEventRecord(startCudaKernelEvent));
	assign<<<N/1024,1024>>>(dA, dB);
	CudaSafeCall(hipEventRecord(endCudaKernelEvent));
	CudaSafeCall(hipEventSynchronize(endCudaKernelEvent));
	CudaSafeCall(hipDeviceSynchronize());
    }
    endCudaKernel = getCurrentTime();
    CudaSafeCall(hipEventElapsedTime(&msecTmp, startCudaKernelEvent, endCudaKernelEvent));
    printf("launch, getCurrentTime = %lf msec, hipEventElapsedTime = %lf msec\n", (float)(endCudaKernel-startCudaKernel)/1000, msecTmp);

    // Step 5: Copy back the data from the device (use cudaMemcpy)
    startCudaMemcpyD2H = getCurrentTime();
    {
	CudaSafeCall(hipEventRecord(startCudaMemcpyD2HEvent));
	CudaSafeCall(hipMemcpy(A, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
	CudaSafeCall(hipEventRecord(endCudaMemcpyD2HEvent));
	CudaSafeCall(hipEventSynchronize(endCudaMemcpyD2HEvent));
    }
    endCudaMemcpyD2H = getCurrentTime();
    CudaSafeCall(hipEventElapsedTime(&msecTmp, startCudaMemcpyD2HEvent, endCudaMemcpyD2HEvent));
    printf("cudaMemcpy, getCurrentTime = %lf msec, hipEventElapsedTime = %lf msec\n", (float)(endCudaMemcpyD2H-startCudaMemcpyD2H)/1000, msecTmp);
    
    // Step 6: Verification
    int error = 0;
    for (int i = 0; i < N; i++) {
	if (A[i] != i) {
	    error++;
	}
    }
    if (!error) {
	printf("VERIFIED\n");
    }
    
    // Step 7: Cleanup
    hipFree(dA);
    hipFree(dB);    
    free(A);
    free(B);
    
    return 0;
}    
