#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

long long getCurrentTime() {
    struct timeval te;
    gettimeofday(&te, NULL); // get current time
    long long microseconds = te.tv_sec*1000000LL + te.tv_usec; 
    return microseconds;
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
	fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		 file, line, hipGetErrorString( err ) );
	exit( -1 );
    }
    #endif

    return;
}

__global__ void assign(int *A, int *B)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    A[i] = B[i];
}

int
main()
{
    int *A, *B;
    int *dA, *dB;
    int N = 256 * 1024 * 1024;
    long long startCudaMalloc, endCudaMalloc;
    long long startCudaMemcpyH2D, endCudaMemcpyH2D;
    long long startCudaKernel, endCudaKernel;
    long long startCudaMemcpyD2H, endCudaMemcpyD2H;
    hipEvent_t startCudaMallocEvent, endCudaMallocEvent;
    hipEvent_t startCudaMemcpyH2DEvent, endCudaMemcpyH2DEvent;
    hipEvent_t startCudaKernelEvent, endCudaKernelEvent;
    hipEvent_t startCudaMemcpyD2HEvent, endCudaMemcpyD2HEvent;
    float msecTmp;

    hipEventCreate(&startCudaMallocEvent);
    hipEventCreate(&endCudaMallocEvent);
    hipEventCreate(&startCudaMemcpyH2DEvent);
    hipEventCreate(&endCudaMemcpyH2DEvent);
    hipEventCreate(&startCudaKernelEvent);
    hipEventCreate(&endCudaKernelEvent);
    hipEventCreate(&startCudaMemcpyD2HEvent);
    hipEventCreate(&endCudaMemcpyD2HEvent);
            
    // Step 1: Allocate memory on the host (use malloc)
    A = (int*)malloc(sizeof(int) * N);
    B = (int*)malloc(sizeof(int) * N);

    for (int i = 0; i < N; i++) {
	B[i] = i;
    }

    // Step 2: Allocate memory on the device (use cudaMalloc)
    startCudaMalloc = getCurrentTime();
    hipEventRecord(startCudaMallocEvent);
    CudaSafeCall(hipMalloc(&dA, sizeof(int) * N));
    CudaSafeCall(hipMalloc(&dB, sizeof(int) * N));
    hipEventRecord(endCudaMallocEvent);
    endCudaMalloc = getCurrentTime();
    hipEventSynchronize(endCudaMallocEvent);
    hipEventElapsedTime(&msecTmp, startCudaMallocEvent, endCudaMallocEvent);
    printf("cudaMalloc, getCurrentTime = %lld usec, hipEventElapsedTime = %lld usec\n", (endCudaMalloc-startCudaMalloc), (long long)(msecTmp*1000));
    
    // Step 3: Copy the host data to the device (use cudaMemcpy) 
    startCudaMemcpyH2D = getCurrentTime();
    hipEventRecord(startCudaMemcpyH2DEvent);
    CudaSafeCall(hipMemcpy(dA, A, sizeof(int) * N, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dB, B, sizeof(int) * N, hipMemcpyHostToDevice));
    hipEventRecord(endCudaMemcpyH2DEvent);
    endCudaMemcpyH2D = getCurrentTime();
    hipEventSynchronize(endCudaMemcpyH2DEvent);
    hipEventElapsedTime(&msecTmp, startCudaMemcpyH2DEvent, endCudaMemcpyH2DEvent);
    printf("cudaMemcpy, getCurrentTime = %lld usec, hipEventElapsedTime = %lld usec\n", (endCudaMemcpyH2D-startCudaMemcpyH2D), (long long)(msecTmp*1000));
    
    // Step 4: Launch the kernel
    startCudaKernel = getCurrentTime();    
    hipEventRecord(startCudaKernelEvent);
    assign<<<N/1024,1024>>>(dA, dB);
    hipEventRecord(endCudaKernelEvent);
    CudaSafeCall(hipGetLastError());
    hipEventSynchronize(endCudaKernelEvent);
    hipDeviceSynchronize();
    endCudaKernel = getCurrentTime();
    hipEventElapsedTime(&msecTmp, startCudaKernelEvent, endCudaKernelEvent);
    printf("launch, getCurrentTime = %lld usec, hipEventElapsedTime = %lld usec\n", (endCudaKernel-startCudaKernel), (long long)(msecTmp*1000));
    
    startCudaMemcpyD2H = getCurrentTime();
    hipEventRecord(startCudaMemcpyD2HEvent);
    // Step 5: Copy back the data from the device (use cudaMemcpy)
    CudaSafeCall(hipMemcpy(A, dA, sizeof(int) * N, hipMemcpyDeviceToHost));
    hipEventRecord(endCudaMemcpyD2HEvent);
    endCudaMemcpyD2H = getCurrentTime();
    hipEventSynchronize(endCudaMemcpyD2HEvent);
    hipEventElapsedTime(&msecTmp, startCudaMemcpyD2HEvent, endCudaMemcpyD2HEvent);
    printf("cudaMemcpy, getCurrentTime = %lld usec, hipEventElapsedTime = %lld usec\n", (endCudaMemcpyD2H-startCudaMemcpyD2H), (long long)(msecTmp*1000));
    
    // Step 6: Verification
    int error = 0;
    for (int i = 0; i < N; i++) {
	if (A[i] != i) {
	    error++;
	}
    }
    if (!error) {
	printf("VERIFIED\n");
    }
    
    // Step 7: Cleanup
    hipFree(dA);
    hipFree(dB);    
    free(A);
    free(B);
    
    return 0;
}    
